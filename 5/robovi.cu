#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 1

__global__ void marginirajGPE(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height)
{
    
// TODO #1.2.1 Določi pixel, ki ga bo nit obdelala
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int levi_rob = 0;
    int desni_rob = width - 1;
    int zgornji_rob = 0;
    int spodnji_rob = height - 1;

// TODO #1.2.2 Določi vrednost vsem sosedom
    int zgoraj = 0;
    int spodaj = 0;
    int levo = 0;
    int desno = 0;
    int zgoraj_levo = 0;
    int zgoraj_desno = 0;
    int spodaj_levo = 0;
    int spodaj_desno = 0;

    if (i != zgornji_rob) {
        zgoraj = imageIn[(i - 1) * width + j];
    }
    if (i != spodnji_rob) {
        spodaj = imageIn[(i + 1) * width + j];
    }
    if (j != levi_rob) {
        levo = imageIn[i * width + j - 1];
    }
    if (j != desni_rob) {
        desno = imageIn[i * width + j + 1];
    }
    if (i != zgornji_rob && j != levi_rob) {
        zgoraj_levo = imageIn[(i - 1) * width + j - 1];
    }
    if (i != zgornji_rob && j != desni_rob) {
        zgoraj_desno = imageIn[(i - 1) * width + j + 1];
    }
    if (i != spodnji_rob && j != levi_rob) {
        spodaj_levo = imageIn[(i + 1) * width + j - 1];
    }
    if (i != spodnji_rob && j != desni_rob) {
        spodaj_desno = imageIn[(i + 1) * width + j + 1];
    }

// TODO #1.2.3 Določi vrednost piksla glede na vrednosti sosedov
    int gx = - zgoraj_levo - 2 * levo - spodaj_levo + zgoraj_desno + 2 * desno + spodaj_desno;
    int gy = zgoraj_levo + 2 * zgoraj + zgoraj_desno - spodaj_levo - 2 * spodaj - spodaj_desno;

    float g = sqrt((float)(gx * gx + gy * gy));

    if (g > 255) {
        g = 255;
    }

    imageOut[i * width + j] = g;
}

void marginirajCPE(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height)
{
    printf("CPE DELA K ZMEŠAN\n");

// TODO #2.2.1 Implementiraj metodo za izvajanje na gostitelju
// TODO #2.2.2 Sprehod po sliki (kot branje - od leve proti desni, od zgoraj navzdol)

    int levi_rob = 0;
    int desni_rob = width - 1;
    int zgornji_rob = 0;
    int spodnji_rob = height - 1;
    
    for (int i = 0; i < width * height; i++) {
        int vrstica = i / width;
        int stolpec = i % width;

// TODO #2.2.3 Določimo vrednost vsem sosedom

        int zgoraj = 0;
        int spodaj = 0;
        int levo = 0;
        int desno = 0;
        int zgoraj_levo = 0;
        int zgoraj_desno = 0;
        int spodaj_levo = 0;
        int spodaj_desno = 0;

// TODO #2.2.4 Preverimo, če smo na robu slike in če nismo, določimo vrednosti pravokotnih sosedov
// TODO #2.2.5 Določimo vrednost vseh osmih sosedov
        if (vrstica != zgornji_rob) {
            zgoraj = imageIn[i - width];
        }
        if (vrstica != spodnji_rob) {
            spodaj = imageIn[i + width];
        }
        if (stolpec != levi_rob) {
            levo = imageIn[i - 1];
        }
        if (stolpec != desni_rob) {
            desno = imageIn[i + 1];
        }
        if (vrstica != zgornji_rob && stolpec != levi_rob) {
            zgoraj_levo = imageIn[i - width - 1];
        }
        if (vrstica != zgornji_rob && stolpec != desni_rob) {
            zgoraj_desno = imageIn[i - width + 1];
        }
        if (vrstica != spodnji_rob && stolpec != levi_rob) {
            spodaj_levo = imageIn[i + width - 1];
        }
        if (vrstica != spodnji_rob && stolpec != desni_rob) {
            spodaj_desno = imageIn[i + width + 1];
        }

// TODO #2.2.6 Določimo vrednost piksla glede na vrednosti sosedov
        int gx = - zgoraj_levo - 2 * levo - spodaj_levo + zgoraj_desno + 2 * desno + spodaj_desno;
        int gy = zgoraj_levo + 2 * zgoraj + zgoraj_desno - spodaj_levo - 2 * spodaj - spodaj_desno;

        double g = sqrt(gx * gx + gy * gy);

        if (g > 255) {
            g = 255;
        }

        imageOut[i] = g;

    }
}

int main(int argc, char *argv[])
{
    printf("------ Comppiled successfully ------\n");
    if (argc < 3)
    {
        printf("USAGE: sample input_image output_image\n");
        exit(EXIT_FAILURE);
    }
    
    char szImage_in_name[255];
    char szImage_out_name[255];
    char szImage_out_nameCPE[255];

    snprintf(szImage_in_name, 255, "./examples/%s", argv[1]);
    snprintf(szImage_out_name, 255, "./resultsGPE/%s", argv[2]);
    snprintf(szImage_out_nameCPE, 255, "./resultsCPE/%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    cpp = COLOR_CHANNELS;

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    // Kot preizkus samo kopiramo vhodno sliko v izhodno
    memcpy(h_imageOut,h_imageIn,datasize);

    // Nastavimo organizacijo niti v 2D
    // dim3 blockSize(1, 1);
    dim3 blockSize(32, 32);
    // dim3 gridSize(1,1);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Rezervacija pomnilnika na napravi
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Uporabimo dogodke CUDA za merjenje casa
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Zazenemo scepec
    hipEventRecord(start);

// TODO #1.1.1 Kopiraj prebrano črno-belo sliko v vhodno sliko na napravi
    checkCudaErrors(hipMemcpy(d_imageIn, h_imageIn, datasize, hipMemcpyHostToDevice));

// TODO #1.1.2 Kliči metodo za izvajanje na napravi
    printf("GPE DELA K ZMEŠAN\n");
    marginirajGPE<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height);
    getLastCudaError("marginirajGPE() execution failed\n");
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    // Izpisemo cas
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Zapisemo izhodno sliko v datoteko
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");

// TODO #2.1.1 Začni meriti čas na gostitelju
    double time_start = clock();

// TODO #2.2.1 Kliči metodo za izvajanje na gostitelju
    marginirajCPE(h_imageIn, h_imageOut, width, height);

// TODO #2.3.1 Nehaj meriti čas na gostitelju in ga izpiši
    double time_end = clock();
    double time_total = (time_end - time_start) / (CLOCKS_PER_SEC / 1000);

    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);
    printf("CPU Execution time is: %0.3f milliseconds \n", time_total);
    printf("Razlika: %0.3f milisekund \n", time_total - milliseconds);
    printf("GPE je porabil %0.3f% časa, ki ga je porabil CPE.\n", (milliseconds / time_total) * 100);

    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }

// TODO #3.1 Glede na filetype izpišemo še rezultat gostiteljskega izvajanja v izhodno datoteko
    if (!strcmp(FileType, "png")){
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
        stbi_write_png(szImage_out_nameCPE, width, height, cpp, h_imageOut, width * cpp);
    }else if (!strcmp(FileType, "jpg")){
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
        stbi_write_jpg(szImage_out_nameCPE, width, height, cpp, h_imageOut, 100);
    }else if (!strcmp(FileType, "bmp")){
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
        stbi_write_bmp(szImage_out_nameCPE, width, height, cpp, h_imageOut);
    }else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Sprostimo pomnilnik na napravi
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Pocistimo dogodke
	hipEventDestroy(start);
	hipEventDestroy(stop);

    // Sprostimo pomnilnik na gostitelju
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}